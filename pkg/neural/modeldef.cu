#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
//CLASSES
//Layer Class
//
#define imin(a,b) (a<b?a:b)
#define ieven(a) ((a%2)==0?true:false)
#define errorChk(a) (a!=hipSuccess?a:hipSuccess)

#define imax(a,b) (a>b?a:b)

#define BLOCK_SIZE 16

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
//
///
//
typedef struct {
  int layer_index; 
  int neurons;
  int prev_neurons;
  bool ready;
  int max;
  //
  float*  cuda_weights; //Temporary vector for loading values to GPU
  float*  cuda_biases;  //Temporary vector for loading values to GPU
  float** weights_by_neuron;     //2D array of sets of weights indexed for each neuron, this makes parallel exec easier
} LayerDefinition;
//Network Class
//
class ModelDefinition
{
private:
  //
  unsigned long model_id;  //lamlearn cloud ID 
  unsigned int layer_num;
  int append_index;
  LayerDefinition* layers;

public: 
  //
  //Constructor
  ModelDefinition(unsigned int model_id,unsigned int layer_num);
  void set_values(unsigned int model_id,unsigned int layer_num);
  void init_layers();
  //
  //Setters
  void append_layer(LayerDefinition layerDef);
  //functions
  //
  //Getters
  unsigned int get_id(){return model_id;};
  int get_append_index(){return append_index;};
  LayerDefinition* get_layer(int id){return &layers[id];}
  LayerDefinition* get_layer_arr_ptr(){return layers;}
  unsigned int get_layer_num(){return layer_num;};
  //
  //dispose
  //
  void dispose(){
    hipFree(layers);
  }
};
//Constructor function
ModelDefinition::ModelDefinition(unsigned int model_id, unsigned int layer_num){
  //
  set_values(model_id, layer_num);
} 
void ModelDefinition::set_values(unsigned int input_model_id, unsigned int input_layer_num){
  //
  model_id  = input_model_id;
  layer_num = input_layer_num;
}
//ModelDefinition Methods
void ModelDefinition::init_layers(){
  //
  // Malloc in managed memory
  hipMallocManaged((void**)&layers, layer_num * sizeof(LayerDefinition));
}
//
void ModelDefinition::append_layer(LayerDefinition layerDef){
  //
  layers[append_index] = layerDef;
  append_index++;
  return;
};
//
//
// device functions
//
__device__ float ReLU(float output){
  //
  if(output < 0){
    return 0;
  } 
  return output;
}
//
__device__ float Sigmoid(float output){
  //
  return output;
}
//
__device__ float Softmax(float output){
  //
  return output;
}
//


__global__ void fast_layer_forward(int N, float* inputs, float *output_buffer, float* weights, float* biases){
  //
  int j = blockIdx.x * blockDim.x + threadIdx.x; // i/max = neuron
  int i = blockIdx.y * blockDim.y + threadIdx.y; // j     = vec pos
  //
  if(j == 0){
    output_buffer[i] = 0.0f;
  } 
  __syncthreads();
  output_buffer[i] += weights[i * N + j]*inputs[j];
  __syncthreads();
  if(j == N-1){
      output_buffer[i] = ReLU(output_buffer[i]+biases[i]);
  }
  //
  return;
}
//
__global__ void fast_4d_forward(int prevN, float* inputs, float *output_buffer, float* weights, float* biases){
  //
  if(threadIdx.x == 0){
    output_buffer[blockDim.x] = 0.0f;
  } 
  __syncthreads();
  output_buffer[blockIdx.x] += weights[(gridDim.x*prevN)+threadIdx.x]*inputs[threadIdx.x];
  __syncthreads();
  if(blockDim.x == threadIdx.x+1){
    output_buffer[blockIdx.x] = ReLU(output_buffer[blockIdx.x]+biases[blockIdx.x]);
  }
  //
  return;
}


void reduce_fraction (int &num, int &den){
  //
  for (int i = den * num; i > 1; i--) {  
    //
    if ((den % i == 0) && (num % i == 0)){ 
      //
      den /= i;  
      num /= i;  
    }      
  }
}

//
//Exported to cGO 
extern "C" {

  int get_max_threads(int device){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    return prop.maxThreadsPerBlock;
  }
  char* get_device_name(int device){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    char *name = (char*)prop.name;
    return name;
  }
  //
  __host__ void execute(int max_threads, char* ptr, float* in, float *out){
    //

    //
    //
    ModelDefinition * model = (ModelDefinition*)ptr;
    int num_layers = model->get_layer_num();
    // printf("num_layers = %i\n", num_layers);
    //Find largest layer && ensure all layers initialized
    int max = 0;
    for(int i = 0; i < num_layers; i++){
      if(max < model->get_layer(i)->neurons){
        max = model->get_layer(i)->neurons;
      }
    }
    //
    // Fire parent kernel
    // 
    float* inputs;
    gpuErrchk( hipMalloc((void**)&inputs, model->get_layer(0)->neurons*sizeof(float)));
    gpuErrchk( hipMemcpy(inputs,in,model->get_layer(0)->neurons*sizeof(float),hipMemcpyHostToDevice));   
    //
    // printf("\n-------EXEC------\n");
    //
    float* output_buffer_even;
    float* output_buffer_odd;
    //
    gpuErrchk( hipMalloc((void**)&output_buffer_even, max*sizeof(float)) ); // Test if it'd be better to init a sized buffer for each layer or stick with a max
    gpuErrchk( hipMalloc((void**)&output_buffer_odd, max*sizeof(float)) ); // Test if it'd be better to init a sized buffer for each layer or stick with a max
    //
    // int start = time_stamp();
    //
    for(int l = 0; l < num_layers; l++){
      //
      // printf("\n- layer %i ----------\n\n", l);
      LayerDefinition* layer = model->get_layer(l);
      //
      //
      int N = layer->neurons;
      int prevN = layer->prev_neurons;
      //
      //
      bool even = ieven(l);
      // 
      if((N*prevN)<=1024){
        //
        //Exec in one pass
        //  
        dim3 threadsPerBlock(N, prevN);
        dim3 numBlocks(1, 1);
        //
        //
        if(l == 0){        //ROOT case 
          //
          fast_layer_forward<<<numBlocks, threadsPerBlock>>>(N, inputs, output_buffer_even, layer->cuda_weights, layer->cuda_biases);
          gpuErrchk( hipPeekAtLastError() );
          gpuErrchk( hipDeviceSynchronize() );
          gpuErrchk( hipFree(inputs) );
          //
        }else if(even){    //EVEN case
          //
          fast_layer_forward<<<numBlocks, threadsPerBlock>>>(N, output_buffer_odd, output_buffer_even, layer->cuda_weights, layer->cuda_biases);
          gpuErrchk( hipPeekAtLastError() );
          gpuErrchk( hipDeviceSynchronize() );
          //
          }else if(!even){ //ODD  case
          //
          fast_layer_forward<<<numBlocks, threadsPerBlock>>>(N, output_buffer_even, output_buffer_odd, layer->cuda_weights, layer->cuda_biases);
          gpuErrchk( hipPeekAtLastError() );
          gpuErrchk( hipDeviceSynchronize() );
          //
        } 
        
      }else{
        //
        //Exec in chunks
        int gridY = 0; 
        int blockX = 0;
        int blockY = 0;
        //
        //
        int n_d = N; 
        int p_d = prevN;
        // printf("N: %i, P: %i\n", N, prevN);
        //
        //
        if(prevN>N||prevN==N){
          //
          n_d = N;
          p_d = prevN/N;
          reduce_fraction(p_d, n_d);
          //
        }else{
          //
          n_d = N;
          p_d = N/prevN;
          reduce_fraction(p_d, n_d);
          //
        }          
        //
          gridY = (prevN/max_threads)+1;
        if(prevN<=max_threads){
          //gridY++;
          blockX = (prevN);
          blockY = 1;
          //
        }else{
          //
          blockX = max_threads/(p_d);
          blockY = (max_threads)/(max_threads/(p_d));
          //
        }
        //
        // printf("n: %i p: %i \nblocksPerGrid(%i, %i)\nthreadsPerBlock(%i, %i)\n", n_d, p_d, N, gridY, blockX, blockY);
        //  
        // blocks per neuron = prevN / 1024
        //
        dim3 numBlocks(N, gridY);
        dim3 threadsPerBlock(blockX, blockY);
        //10240 inputs = 10 blocks per neuron
        // 

        //
        //
        // hipError_t err;
        if(l == 0){     //ROOT case 
          fast_4d_forward<<<numBlocks, threadsPerBlock>>>(prevN, inputs, output_buffer_even, layer->cuda_weights, layer->cuda_biases);
          gpuErrchk( hipPeekAtLastError() );
          gpuErrchk( hipDeviceSynchronize() );
          gpuErrchk( hipFree(inputs) );
        }else if(even){ //EVEN case
          fast_4d_forward<<<numBlocks, threadsPerBlock>>>(prevN, output_buffer_odd, output_buffer_even, layer->cuda_weights, layer->cuda_biases);
          gpuErrchk( hipPeekAtLastError() );
          gpuErrchk( hipDeviceSynchronize() );
        }else{          //ODD  case
          fast_4d_forward<<<numBlocks, threadsPerBlock>>>(prevN, output_buffer_even, output_buffer_odd, layer->cuda_weights, layer->cuda_biases);
          gpuErrchk( hipPeekAtLastError() );
          gpuErrchk( hipDeviceSynchronize() );
        } 
      }
      //
      // copy last layer output
      if(l == num_layers-1){
        if(even){ 
          //EVEN case
          gpuErrchk(  hipMemcpy(out,output_buffer_even,layer->neurons*sizeof(float),hipMemcpyDeviceToHost) );
        }else{    
          //ODD  case
          gpuErrchk(  hipMemcpy(out,output_buffer_odd,layer->neurons*sizeof(float),hipMemcpyDeviceToHost) );
          
        } 
      }
      //
      // 
      //

    } // End layers
    gpuErrchk( hipFree(output_buffer_odd) );
    gpuErrchk( hipFree(output_buffer_even) );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    //
    return;
  }




  //Model permutation methods

  char* newModel(int ModelLibraryID, int LayerNum){
    //Create a new model
    ModelDefinition* model = new(ModelDefinition)(ModelLibraryID,LayerNum);
    model->init_layers();
    return (char*)model;

  }

  int appendLayer(char* ptr, const int N, const int prevN, float* weights, float* biases){
    //   
    // Cast ptr to model
    ModelDefinition* modelDefPtr = (ModelDefinition*)ptr;
    //
    LayerDefinition layerDef = LayerDefinition();
    layerDef.layer_index = modelDefPtr->get_append_index();
    layerDef.neurons = N; 
    layerDef.ready = false;
    layerDef.prev_neurons = prevN; 
    // //
    //GPU alloc sizes
    size_t weight_size = (N*prevN)* sizeof(float);
    size_t biases_size = N * sizeof(float);
    //
    //Alloc on GPU
    gpuErrchk(hipMalloc((void**)&layerDef.cuda_weights, weight_size));
    gpuErrchk(hipMemcpy(layerDef.cuda_weights,weights,weight_size,hipMemcpyHostToDevice));
    //
    gpuErrchk(hipMalloc((void**)&layerDef.cuda_biases, biases_size));
    gpuErrchk(hipMemcpy(layerDef.cuda_biases,biases,biases_size,hipMemcpyHostToDevice));
    //
    //
    modelDefPtr->append_layer(layerDef);
    return 0;
  }



  // Model data methods

  int getModelID(char* ptr){
    //Append new layer to previous layer of model
    ModelDefinition* modelPtr = (ModelDefinition*)ptr;
    return (int)modelPtr->get_id();
  }


  void disposeModel(char* ptr){
    //
    ModelDefinition* modelPtr = (ModelDefinition*)ptr;
    modelPtr->dispose();
    return;
  }

  // Device Methods

  void printCudaInfo(void){
    int nDevices;
    // Use this code later ;) 
    hipGetDeviceCount(&nDevices);
    //
    for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      
      hipGetDeviceProperties(&prop, i);
      printf("Device Number: %d\n", i);
      printf("  Max Threads: %i\n", prop.maxThreadsPerBlock);
      printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
      printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
      printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
  }
}

